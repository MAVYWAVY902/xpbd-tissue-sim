#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include <chrono>
#include <thread>

#include "utils/CudaHelperMath.h"

#include "common/types.hpp"

#include "utils/MeshUtils.hpp"
#include "geometry/Mesh.hpp"
#include "geometry/SphereSDF.hpp"
#include "geometry/CylinderSDF.hpp"
#include "simobject/RigidPrimitives.hpp"
#include "gpu/GPUStructs.hpp"
#include "gpu/resource/GPUResource.hpp"
#include "gpu/resource/SphereSDFGPUResource.hpp"
#include "gpu/resource/BoxSDFGPUResource.hpp"
#include "gpu/resource/MeshGPUResource.hpp"
#include "gpu/resource/CylinderSDFGPUResource.hpp"
#include "gpu/resource/WritableArrayGPUResource.hpp"

#include "config/RigidPrimitiveConfigs.hpp"

struct GPUCollision
{
    float penetration_dist;
    float3 bary_coords;
};

__device__ float sphere_sdf_distance(const Sim::GPUSphereSDF* sphere_sdf, const float3& x)
{
    return length(x - sphere_sdf->position) - sphere_sdf->radius;
}

__device__ float3 sphere_sdf_gradient(const Sim::GPUSphereSDF* sphere_sdf, const float3& x)
{
    float3 grad =  normalize(x - sphere_sdf->position);
    // printf("x: %f, %f, %f\n", x.x, x.y, x.z);
    // printf("grad: %f, %f, %f\n", grad.x, grad.y, grad.z);
    return grad;
}

__global__ void sphereMeshCollisionDetection(const Sim::GPUSphereSDF* sphere_sdf, const float* vertices, int num_vertices, const int* faces, int num_faces, GPUCollision* collisions)
{
    int face_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (face_index >= num_faces)    return;
    const int* face = faces + 3*face_index;
    float3 x1, x2, x3;
    x1.x = (vertices + 3*face[0])[0];  x1.y = (vertices + 3*face[0])[1]; x1.z = (vertices + 3*face[0])[2];
    x2.x = (vertices + 3*face[1])[0];  x2.y = (vertices + 3*face[1])[1]; x2.z = (vertices + 3*face[1])[2];
    x3.x = (vertices + 3*face[2])[0];  x3.y = (vertices + 3*face[2])[1]; x3.z = (vertices + 3*face[2])[2];

    // printf("f: %i, %i, %i\n", faces[0], faces[1], faces[2]);
    // printf("x1: %f, %f, %f\n", x1.x, x1.y, x1.z);
    // printf("x2: %f, %f, %f\n", x2.x, x2.y, x2.z);
    // printf("x3: %f, %f, %f\n", x3.x, x3.y, x3.z);
    // printf("x1: %f, %f, %f\n", vertices[0], vertices[1], vertices[2]);
    // printf("x2: %f, %f, %f\n", vertices[3], vertices[4], vertices[5]);
    // printf("x3: %f, %f, %f\n", vertices[6], vertices[7], vertices[8]);

    float3 min_p = x1;
    float3 min_bary_coords = make_float3(1.0, 0.0, 0.0);
    float min_dist = 1000;

    const int num_samples = 16;
    for (int i = 0; i <= num_samples; i++)
    {
        for (int j = 0; j <= num_samples - i; j++)
        {
            const float u = (float)i / num_samples;
            const float v = (float)j / num_samples;
            const float w = 1 - u - v;
            float3 p = u*x1 + v*x2 + w*x3;
            // printf("u: %f, v: %f, w: %f ", u, v, w);
            // printf("p: %f,%f,%f\n", p.x, p.y, p.z);
            const float dist = sphere_sdf_distance(sphere_sdf, p);

            if (dist < min_dist)
            {
                min_dist = dist;
                min_p = p;
                min_bary_coords = make_float3(u, v, w);
            }
        }
    }

    // printf("d: %f\n", min_dist);
    // printf("f: %i min_p: %f, %f, %f\n", face_index, min_p.x, min_p.y, min_p.z);
    // TODO: stream compaction
    // float3 grad = sphere_sdf_gradient(sphere_sdf, min_p);
    collisions[face_index].penetration_dist = min_dist;
    // collisions[face_index].normal = sphere_sdf_gradient(sphere_sdf, min_p);
    collisions[face_index].bary_coords = min_bary_coords;
    // collisions[face_index].surface_point = min_p - collisions[face_index].normal * min_dist;
    
}

__global__ void sphereMeshCollisionDetectionParallel(const Sim::GPUSphereSDF* sphere_sdf, const float* vertices, int num_vertices, const int* faces, int num_faces, GPUCollision* collisions)
{
    int face_index = blockIdx.x;
    __shared__ int3 face;
    face = make_int3(faces[3*face_index], faces[3*face_index+1], faces[3*face_index+2]);
    __shared__ float3 x1, x2, x3;
    x1.x = (vertices + 3*face.x)[0];  x1.y = (vertices + 3*face.x)[1]; x1.z = (vertices + 3*face.x)[2];
    x2.x = (vertices + 3*face.y)[0];  x2.y = (vertices + 3*face.y)[1]; x2.z = (vertices + 3*face.y)[2];
    x3.x = (vertices + 3*face.z)[0];  x3.y = (vertices + 3*face.z)[1]; x3.z = (vertices + 3*face.z)[2];
    // const float* x1 = vertices + 3*face[0];
    // const float* x2 = vertices + 3*face[1];
    // const float* x3 = vertices + 3*face[2];

    // map thread index to iteration
    const int N = 22;
    const int n = ceilf(0.5 * (2*N + 1) - sqrtf( (2*N + 1)*(2*N + 1) - 8*(threadIdx.x + 1)));
    const int i = min(n - 1, N - 1);    // clamp i to be still in the triangle
    const int j = threadIdx.x - (2*N - i + 1) * i / 2;

    // calculate barycentric coords u,v,w
    const float u = (float)i / (N - 1);
    const float v = (float)j / (N - 1);
    const float w = 1 - u - v;

    // find the distance
    float3 p = u*x1 + v*x2 + w*x3;
    const float dist = sphere_sdf_distance(sphere_sdf, p);
}

__device__ void global_to_body(const float3& x, const float3& body_position, const float4& body_orientation, float3& x_body)
{
    const float3 diff = x - body_position;
    const float3 neg_quat3 = make_float3(-body_orientation.x, -body_orientation.y, -body_orientation.z);
    x_body = 2*dot(diff,neg_quat3)*neg_quat3 + (2*body_orientation.w*body_orientation.w - 1)*diff + 2*body_orientation.w*cross(neg_quat3, diff);
}

__device__ float cyl_sdf_distance(const Sim::GPUCylinderSDF* cyl_sdf, const float3& x)
{
    float3 x_body;
    global_to_body(x, cyl_sdf->position, cyl_sdf->orientation, x_body);

    const float xy_dist = sqrtf(x_body.x*x_body.x + x_body.y*x_body.y) - cyl_sdf->radius;
    const float z_dist = fabs(x_body.z) - 0.5 * cyl_sdf->height;

    const float outside_dist_xy = fmaxf(xy_dist, 0.0);
    const float outside_dist_z = fmaxf(z_dist, 0.0);

    const float dist_when_outside = sqrtf(outside_dist_xy*outside_dist_xy + outside_dist_z*outside_dist_z);
    const float dist_when_inside = fminf(fmaxf(xy_dist, z_dist), 0.0);

    return dist_when_inside + dist_when_outside;
}

__global__ void cylinderMeshCollisionDetection(const Sim::GPUCylinderSDF* cyl_sdf, const float* vertices, int num_vertices, const int* faces, int num_faces, GPUCollision* collisions)
{
    int face_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (face_index >= num_faces)    return;
    const int* face = faces + 3*face_index;
    float3 x1, x2, x3;
    x1.x = (vertices + 3*face[0])[0];  x1.y = (vertices + 3*face[0])[1]; x1.z = (vertices + 3*face[0])[2];
    x2.x = (vertices + 3*face[1])[0];  x2.y = (vertices + 3*face[1])[1]; x2.z = (vertices + 3*face[1])[2];
    x3.x = (vertices + 3*face[2])[0];  x3.y = (vertices + 3*face[2])[1]; x3.z = (vertices + 3*face[2])[2];

    float3 min_p = x1;
    float3 min_bary_coords = make_float3(1.0, 0.0, 0.0);
    float min_dist = 1000;

    const int num_samples = 16;
    for (int i = 0; i <= num_samples; i++)
    {
        for (int j = 0; j <= num_samples - i; j++)
        {
            const float u = (float)i / num_samples;
            const float v = (float)j / num_samples;
            const float w = 1 - u - v;
            float3 p = u*x1 + v*x2 + w*x3;
            const float dist = cyl_sdf_distance(cyl_sdf, p);

            if (dist < min_dist)
            {
                min_dist = dist;
                min_p = p;
                min_bary_coords = make_float3(u, v, w);
            }
        }
    }

    // TODO: stream compaction
    collisions[face_index].penetration_dist = min_dist;
    collisions[face_index].bary_coords = min_bary_coords;
}


__host__ void launchCollisionKernel(const Sim::HostReadableGPUResource* sdf_resource, const Sim::MeshGPUResource* mesh_resource, int num_vertices, int num_faces, Sim::WritableArrayGPUResource<GPUCollision>* collisions_resource)
{
    const int block_size = 256;
    const int num_blocks = (num_faces + block_size - 1) / block_size;
    // const int num_blocks = num_faces;

    // spawn GPU kernel depending on the type of SDF being collided with
    if (const Sim::SphereSDFGPUResource* sphere_sdf_resource = dynamic_cast<const Sim::SphereSDFGPUResource*>(sdf_resource))
    {
        sphereMeshCollisionDetection<<<num_blocks, block_size>>>(sphere_sdf_resource->gpuSDF(),
                                                                 mesh_resource->gpuVertices(),
                                                                 num_vertices,
                                                                 mesh_resource->gpuFaces(),
                                                                 num_faces,
                                                                 collisions_resource->gpuArr());
        // sphereMeshCollisionDetectionParallel<<<num_blocks, block_size>>>(sphere_sdf_resource->gpuSDF(),
        //                                                          mesh_resource->gpuVertices(),
        //                                                          num_vertices,
        //                                                          mesh_resource->gpuFaces(),
        //                                                          num_faces,
        //                                                          collision_resource->gpuArr());
        CHECK_CUDA_ERROR(hipPeekAtLastError());
        // remove later, but here for testing
        // CHECK_CUDA_ERROR(hipDeviceSynchronize());
    }
    else if (const Sim::BoxSDFGPUResource* box_sdf_resource = dynamic_cast<const Sim::BoxSDFGPUResource*>(sdf_resource))
    {

    }
    else if (const Sim::CylinderSDFGPUResource* cyl_sdf_resource = dynamic_cast<const Sim::CylinderSDFGPUResource*>(sdf_resource))
    {
        cylinderMeshCollisionDetection<<<num_blocks, block_size>>>(cyl_sdf_resource->gpuSDF(),
                                                                    mesh_resource->gpuVertices(),
                                                                    num_vertices,
                                                                    mesh_resource->gpuFaces(),
                                                                    num_faces,
                                                                    collisions_resource->gpuArr());
    }

    
}

int main(void)
{
    gmsh::initialize();

    Geometry::TetMesh mesh = MeshUtils::loadTetMeshFromGmshFile("../resource/cube/cube8.msh");
    std::cout << "NUM FACES: " << mesh.numFaces() << std::endl;
    mesh.resize(1.0);
    mesh.moveTogether(Vec3r(-0.5, -0.5, 0.499));
    // mesh.moveTogether(Vec3r(0, 0, 0.5));
    mesh.createGPUResource();
    const Sim::MeshGPUResource* mesh_gpu_resource = dynamic_cast<const Sim::MeshGPUResource*>(mesh.gpuResource());
    mesh_gpu_resource->fullCopyToDevice();

    // RigidSphereConfig sphere_config("sphere1", Vec3r(0,0,0), Vec3r(0,0,0), Vec3r(0,0,0), Vec3r(0,0,0), 100, 0.5, true, true);
    // Sim::RigidSphere sphere(nullptr, &sphere_config);
    // Geometry::SphereSDF sdf(&sphere);
    RigidCylinderConfig cyl_config("cyl1", Vec3r(0,0,0), Vec3r(0,0,0), Vec3r(0,0,0), Vec3r(0,0,0), 100, 0.5, 1, true, true);
    Sim::RigidCylinder cyl(nullptr, &cyl_config);
    Geometry::CylinderSDF sdf(&cyl);
    sdf.createGPUResource();
    sdf.gpuResource()->fullCopyToDevice();

    GPUCollision* collisions = new GPUCollision[mesh.numFaces()];
    std::unique_ptr<Sim::WritableArrayGPUResource<GPUCollision>> collisions_resource = std::make_unique<Sim::WritableArrayGPUResource<GPUCollision>>(collisions, mesh.numFaces());
    collisions_resource->allocate();

    // allocate SDF on device
    int block_size = 256;
    int num_blocks;
    num_blocks = (mesh.numFaces() + block_size - 1) / block_size;

    // warm up
    launchCollisionKernel(sdf.gpuResource(), mesh_gpu_resource, mesh.numVertices(), mesh.numFaces(), collisions_resource.get());

    hipDeviceSynchronize();

    std::array<int, 100> nanosecs;
    for (int i = 0; i < 100; i++)
    {
        // std::this_thread::sleep_for(std::chrono::milliseconds(100));
        /////////////////////////////////////////////////////////////////////
        auto start2 = std::chrono::high_resolution_clock::now();

        sdf.gpuResource()->partialCopyToDevice();
        mesh_gpu_resource->partialCopyToDevice();
        

        launchCollisionKernel(sdf.gpuResource(), mesh_gpu_resource, mesh.numVertices(), mesh.numFaces(), collisions_resource.get());

        // hipDeviceSynchronize();

        collisions_resource->copyFromDevice();
        

        auto end2 = std::chrono::high_resolution_clock::now();
        auto nanosec2 = std::chrono::duration_cast<std::chrono::nanoseconds>(end2 - start2);
        // std::cout << "Elapsed time on GPU: " << nanosec2.count() << " ns" << std::endl;
        nanosecs[i] = nanosec2.count();
        /////////////////////////////////////////////////////////////////////
    }
    for (const auto& t : nanosecs)
    {
        std::cout << "Elapsed time on GPU: " << t << " ns\n";
    }

    // for (int i = 0; i < mesh.numFaces(); i++)
    // {
    //     if (collisions[i].penetration_dist < 0)
    //         std::cout << "dist " << i << ": " << collisions[i].penetration_dist << std::endl;
    // }
    // auto start = std::chrono::high_resolution_clock::now();

    // num_blocks = num_faces;
    // sphereMeshCollisionDetectionParallel<<<num_blocks, block_size>>>(d_sdf, d_vertices, num_vertices, d_faces, num_faces);

    // // wait for GPU
    // hipDeviceSynchronize();

    // auto end = std::chrono::high_resolution_clock::now();
    // auto nanosec = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
    // std::cout << "Elapsed time on GPU (more parallel): " << nanosec.count() << " ns" << std::endl;

    ///////////////////////////////////////////////////////////////////

    for (int i = 0; i < 3; i++)
    {
    auto start3 = std::chrono::high_resolution_clock::now();
    for (int face_index = 0; face_index < mesh.numFaces(); face_index++)
    {
        const Vec3i face = mesh.face(face_index);
        const Vec3r x1 = mesh.vertex(face[0]);
        const Vec3r x2 = mesh.vertex(face[1]);
        const Vec3r x3 = mesh.vertex(face[2]);
        // const float* x1 = mesh.vertices().data() + 3*face[0];
        // const float* x2 = mesh.vertices().data() + 3*face[1];
        // const float* x3 = mesh.vertices().data() + 3*face[2];

        Vec3r min_bary_coords(1, 0, 0);
        Vec3r min_p = x1;
        float min_dist = 1000;

        const int num_samples = 16;
        for (int i = 0; i <= num_samples; i++)
        {
            for (int j = 0; j <= num_samples; j++)
            {
                const float u = (float)i / num_samples;
                const float v = (float)j / num_samples;
                const float w = 1 - u - v;
                const Vec3r p = u*x1 + v*x2 + w*x3;
                const float dist = sdf.evaluate(p);
                
                if (dist < min_dist)
                {
                    min_dist = dist;
                    min_p = p;
                    min_bary_coords = Vec3r(u,v,w);
                }

            }
        }
    }
    auto end3 = std::chrono::high_resolution_clock::now();
    auto nanosec3 = std::chrono::duration_cast<std::chrono::nanoseconds>(end3 - start3);
    std::cout << "Elapsed time on CPU: " << nanosec3.count()/1000 << " us" << std::endl;
    }
    delete[] collisions;
    

    return 0;
}