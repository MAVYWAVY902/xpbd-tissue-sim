#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include <chrono>
#include <thread>

#include "utils/CudaHelperMath.h"

#include "common/types.hpp"

#include "utils/MeshUtils.hpp"
#include "geometry/Mesh.hpp"
#include "geometry/SphereSDF.hpp"
#include "simobject/XPBDMeshObject.hpp"
#include "config/XPBDMeshObjectConfig.hpp"

#include "gpu/GPUStructs.hpp"
#include "gpu/GPUResource.hpp"
#include "gpu/TetMeshGPUResource.hpp"
#include "gpu/ArrayGPUResource.hpp"
#include "gpu/WritableArrayGPUResource.hpp"

// computes A * B^T and stores result in C
__host__ __device__ void Mat3MulTranspose(const float* A, const float* B, float* C)
{
    C[0] = A[0]*B[0] + A[3]*B[3] + A[6]*B[6];
    C[1] = A[1]*B[0] + A[4]*B[3] + A[7]*B[6];
    C[2] = A[2]*B[0] + A[5]*B[3] + A[8]*B[6];

    C[3] = A[0]*B[1] + A[3]*B[4] + A[6]*B[7];
    C[4] = A[1]*B[1] + A[4]*B[4] + A[7]*B[7];
    C[5] = A[2]*B[1] + A[5]*B[4] + A[8]*B[7];

    C[6] = A[0]*B[2] + A[3]*B[5] + A[6]*B[8];
    C[7] = A[1]*B[2] + A[4]*B[5] + A[7]*B[8];
    C[8] = A[2]*B[2] + A[5]*B[5] + A[8]*B[8];
}

// computes A * B and stores result in C
__host__ __device__ void Mat3Mul(const float* A, const float* B, float* C)
{
    C[0] = A[0]*B[0] + A[3]*B[1] + A[6]*B[2];
    C[1] = A[1]*B[0] + A[4]*B[1] + A[7]*B[2];
    C[2] = A[2]*B[0] + A[5]*B[1] + A[8]*B[2];

    C[3] = A[0]*B[3] + A[3]*B[4] + A[6]*B[5];
    C[4] = A[1]*B[3] + A[4]*B[4] + A[7]*B[5];
    C[5] = A[2]*B[3] + A[5]*B[4] + A[8]*B[5];

    C[6] = A[0]*B[6] + A[3]*B[7] + A[6]*B[8];
    C[7] = A[1]*B[6] + A[4]*B[7] + A[7]*B[8];
    C[8] = A[2]*B[6] + A[5]*B[7] + A[8]*B[8];
}

__host__ __device__ void Vec3Cross(const float* v1, const float* v2, float* v3)
{
    v3[0] = v1[1]*v2[2] - v1[2]*v2[1];
    v3[1] = v1[2]*v2[0] - v1[0]*v2[2];
    v3[2] = v1[0]*v2[1] - v1[1]*v2[0];
}

__host__ __device__ void ElementJacobiSolve(int elem_index, const int* elements, int num_elements, const float* vertices, const float* masses, const float* volumes, const float* Qs, float lambda, float mu, float dt, float* coord_updates)
{
    // extract quantities for element
    const int* elem = elements + 4*elem_index;

    float3 x1, x2, x3, x4;
    x1.x = (vertices + 3*elem[0])[0];  x1.y = (vertices + 3*elem[0])[1]; x1.z = (vertices + 3*elem[0])[2];
    x2.x = (vertices + 3*elem[1])[0];  x2.y = (vertices + 3*elem[1])[1]; x2.z = (vertices + 3*elem[1])[2];
    x3.x = (vertices + 3*elem[2])[0];  x3.y = (vertices + 3*elem[2])[1]; x3.z = (vertices + 3*elem[2])[2];
    x4.x = (vertices + 3*elem[3])[0];  x4.y = (vertices + 3*elem[3])[1]; x4.z = (vertices + 3*elem[3])[2];

    // float inv_m1 = 1.0/masses[3*elem[0]];
    // float inv_m2 = 1.0/masses[3*elem[1]];
    // float inv_m3 = 1.0/masses[3*elem[2]];
    // float inv_m4 = 1.0/masses[3*elem[3]];

    float inv_m[4];
    for (int i = 0; i < 4; i++) { inv_m[i] = 1.0/masses[3*elem[i]]; }

    const float* Q = Qs + 9*elem_index;
    const float gamma = mu / lambda;

    // compute F
    float X[9];
    X[0] = x1.x - x4.x; X[1] = x1.y - x4.y; X[2] = x1.z - x4.z;
    X[3] = x2.x - x4.x; X[4] = x2.y - x4.y; X[5] = x2.z - x4.z;
    X[6] = x3.x - x4.x; X[7] = x3.y - x4.y; X[8] = x3.z - x4.z;

    float F[9];
    Mat3Mul(X, Q, F);

    // compute hydrostatic constraint and its gradient
    float C_h_grad[12];

    // C_h = det(F) - (1 + gamma)
    const float C_h = F[0]*F[4]*F[8] - F[0]*F[7]*F[5] - F[3]*F[1]*F[8] + F[3]*F[7]*F[2] + F[6]*F[1]*F[5] - F[6]*F[4]*F[2] - (1+gamma);

    const float alpha_h = 1.0/(lambda * volumes[elem_index]);

    float F_cross[9];
    Vec3Cross(F+3, F+6, F_cross);   // 2nd column of F crossed with 3rd column
    Vec3Cross(F+6, F, F_cross+3);   // 3rd column of F crossed with 1st column
    Vec3Cross(F, F+3, F_cross+6);   // 1st column of F crossed with 2nd column

    Mat3MulTranspose(F_cross, Q, C_h_grad);
    C_h_grad[9] = -C_h_grad[0] - C_h_grad[3] - C_h_grad[6];
    C_h_grad[10] = -C_h_grad[1] - C_h_grad[4] - C_h_grad[7];
    C_h_grad[11] = -C_h_grad[2] - C_h_grad[5] - C_h_grad[8];


    // compute deviatoric constraint and its gradient
    float C_d_grad[12];

    // C_d = frob(F)
    const float C_d = sqrtf(F[0]*F[0] + F[1]*F[1] + F[2]*F[2] + F[3]*F[3] + F[4]*F[4] + F[5]*F[5] + F[6]*F[6] + F[7]*F[7] + F[8]*F[8]);
    const float inv_C_d = 1.0/C_d;

    const float alpha_d = 1.0/(mu * volumes[elem_index]);

    Mat3MulTranspose(F, Q, C_d_grad);
    for (int i = 0; i < 9; i++) { C_d_grad[i] *= inv_C_d; }
    C_d_grad[9] =  -C_d_grad[0] - C_d_grad[3] - C_d_grad[6];
    C_d_grad[10] = -C_d_grad[1] - C_d_grad[4] - C_d_grad[7];
    C_d_grad[11] = -C_d_grad[2] - C_d_grad[5] - C_d_grad[8];


    // solve the 2x2 system
    float A[4];
    A[0] = alpha_h / (dt * dt);
    A[1] = 0;
    A[2] = 0;
    A[3] = alpha_d / (dt * dt);
    for (int i = 0; i < 4; i++)
    {
        A[0] += inv_m[i] * (C_h_grad[3*i]*C_h_grad[3*i] + C_h_grad[3*i+1]*C_h_grad[3*i+1] + C_h_grad[3*i+2]*C_h_grad[3*i+2]);
        A[3] += inv_m[i] * (C_d_grad[3*i]*C_d_grad[3*i] + C_d_grad[3*i+1]*C_d_grad[3*i+1] + C_d_grad[3*i+2]*C_d_grad[3*i+2]);
        A[1] += inv_m[i] * (C_h_grad[3*i]*C_d_grad[3*i] + C_h_grad[3*i+1]*C_d_grad[3*i+1] + C_h_grad[3*i+2]*C_d_grad[3*i+2]);
    }
    A[2] = A[1];

    float k[2];
    k[0] = -C_h; // TODO: include current lambda (assuming its 0 right now)
    k[1] = -C_d; // TODO: include current lambda

    const float detA = A[0]*A[3] - A[1]*A[2];
    const float dlam_h = (k[0]*A[3] - k[1]*A[2]) / detA;
    const float dlam_d = (k[1]*A[0] - k[0]*A[1]) / detA;

    // compute the coordinate updates
    for (int i = 0; i < 4; i++)
    {
        coord_updates[12*elem_index + 3*i] =   inv_m[i] * (C_h_grad[3*i] * dlam_h + C_d_grad[3*i] * dlam_d);
        coord_updates[12*elem_index + 3*i+1] = inv_m[i] * (C_h_grad[3*i+1] * dlam_h + C_d_grad[3*i+1] * dlam_d);
        coord_updates[12*elem_index + 3*i+2] = inv_m[i] * (C_h_grad[3*i+2] * dlam_h + C_d_grad[3*i+2] * dlam_d);
    }
}

__global__ void XPBDJacobiSolve(const int* elements, int num_elements, const float* vertices, const float* masses, const float* volumes, const float* Qs, float lambda, float mu, float dt, float* coord_updates)
{
    int elem_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (elem_index >= num_elements)    return;

    for (int gi = 0; gi < 1; gi++)
    {
        ElementJacobiSolve(elem_index, elements, num_elements, vertices, masses, volumes, Qs, lambda, mu, dt, coord_updates);
    }

}

int main(void)
{
    gmsh::initialize();

    // MeshUtils::createBeamObj("../resource/cube/cube32.obj", 1, 1, 1, 32);
    // MeshUtils::convertToSTL("../resource/cube/cube32.obj");
    // MeshUtils::convertSTLtoMSH("../resource/cube/cube32.stl");

    Geometry::TetMesh mesh = MeshUtils::loadTetMeshFromGmshFile("../resource/cube/cube16.msh");
    mesh.resize(1.0);
    mesh.moveTogether(Vec3r(-0.5, -0.5, 0.499));
    // mesh.moveTogether(Vec3r(0, 0, 0.5));
    mesh.createGPUResource();
    const Sim::TetMeshGPUResource* mesh_gpu_resource = dynamic_cast<const Sim::TetMeshGPUResource*>(mesh.gpuResource());
    mesh_gpu_resource->fullCopyToDevice();


    float density = 1000;
    float lambda = 1e7;
    float mu = 1e6;

    // precompute quantities
    std::vector<float> volumes(mesh.numElements());
    std::vector<float> Qs(mesh.numElements()*9);
    std::vector<float> masses(mesh.numVertices());
    for (int i = 0; i < mesh.numElements(); i++)
    {
        const Eigen::Vector4i elem = mesh.element(i);
        Eigen::Matrix3f X;
        X.col(0) = mesh.vertex(elem[0]) - mesh.vertex(elem[3]);
        X.col(1) = mesh.vertex(elem[1]) - mesh.vertex(elem[3]);
        X.col(2) = mesh.vertex(elem[2]) - mesh.vertex(elem[3]);

        Eigen::Matrix3f Q = X.inverse();
        for (int k = 0; k < 9; k++) { Qs[9*i+k] = Q.data()[k]; }

        volumes[i] = std::abs(X.determinant()/6.0);

        float element_mass = volumes[i] * density;
        for (int k = 0; k < 4; k++) { masses[elem[k]] += element_mass / 4.0; }
    }

    Sim::ArrayGPUResource<float> volumes_resource(volumes.data(), volumes.size());
    volumes_resource.allocate();
    volumes_resource.fullCopyToDevice();
    Sim::ArrayGPUResource<float> Qs_resource(Qs.data(), Qs.size());
    Qs_resource.allocate();
    Qs_resource.fullCopyToDevice();
    Sim::ArrayGPUResource<float> masses_resource(masses.data(), masses.size());
    masses_resource.allocate();
    masses_resource.fullCopyToDevice();

    std::vector<float> coord_updates(mesh.numElements()*12);
    Sim::WritableArrayGPUResource<float> coord_updates_resource(coord_updates.data(), coord_updates.size());
    coord_updates_resource.allocate();

    // allocate SDF on device
    int block_size = 256;
    int num_blocks;
    num_blocks = (mesh.numElements() + block_size - 1) / block_size;

    std::array<int, 100> nanosecs;
    for (int i = 0; i < 100; i++)
    {
        // std::this_thread::sleep_for(std::chrono::milliseconds(100));
        /////////////////////////////////////////////////////////////////////
        auto start2 = std::chrono::high_resolution_clock::now();

        mesh_gpu_resource->partialCopyToDevice();
        

        XPBDJacobiSolve<<<num_blocks, block_size>>>(mesh_gpu_resource->gpuElements(), mesh.numElements(), mesh_gpu_resource->gpuVertices(),
                                                     masses_resource.gpuArr(), volumes_resource.gpuArr(), Qs_resource.gpuArr(),
                                                    lambda, mu, 1e-3,
                                                    coord_updates_resource.gpuArr());
        CHECK_CUDA_ERROR(hipPeekAtLastError());

        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        

        auto end2 = std::chrono::high_resolution_clock::now();
        auto nanosec2 = std::chrono::duration_cast<std::chrono::nanoseconds>(end2 - start2);
        // std::cout << "Elapsed time on GPU: " << nanosec2.count() << " ns" << std::endl;
        nanosecs[i] = nanosec2.count();
        /////////////////////////////////////////////////////////////////////
    }
    for (const auto& t : nanosecs)
    {
        std::cout << "Elapsed time on GPU: " << t << " ns\n";
    }

    /////////////////////////////////////////////////////////////////////////

    auto start3 = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < mesh.numElements(); i++)
    {
        ElementJacobiSolve(i, mesh.elements().data(), mesh.numElements(), mesh.vertices().data(),
                            masses.data(), volumes.data(), Qs.data(),
                            lambda, mu, 1e-3,
                            coord_updates.data());
    }
    auto end3 = std::chrono::high_resolution_clock::now();
    auto nanosec3 = std::chrono::duration_cast<std::chrono::nanoseconds>(end3 - start3);
    std::cout << "Elapsed time on CPU: " << nanosec3.count()/1000000 << " ms" << std::endl;

    ///////////////////////////////////////////////////////////////////
    

    return 0;
}